#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>
#include <thread>
#include <cstring>
#include <cstdio>

int main()
{
    void *dst, *src;
    auto size = 32;
    hipMalloc(&dst, size);
    hipHostMalloc(&src, size, hipHostMallocDefault);
    unsigned long times = 256ul * 256 * 256 * 3;
    for (unsigned long i = 0; i < times; i++) {
        hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice);
    }
    return 0;
}